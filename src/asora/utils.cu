#include "utils.cuh"

#include <exception>
#include <format>

namespace asora {

    void safe_cuda(hipError_t err, const std::source_location &loc) {
        if (err != hipSuccess)
            throw std::runtime_error(
                std::format("CUDA Error {}: {}. At {} in {}:{}", hipGetErrorName(err),
                            hipGetErrorString(err), loc.function_name(),
                            loc.file_name(), loc.line()));
    }

}  // namespace asora
